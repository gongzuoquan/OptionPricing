#include<stdio.h>
#include"hip/hip_runtime.h"
#include"nccl.h"
#include"option.hpp"
#include<iostream>
#define DNUM 2
void test_option(int simN,int time_step)
{
    auto start=system_clock::now();
    //dtype S0=100,K=100,T=0.5,V=0.5,R=0.1;
    //dtype S0=164,K=166,T=0.0959,V=0.29,R=0.0521;

    //dtype S0=50,K=60,T=1,V=0.4,R=0.1;
    dtype S0=100,K=100,T=1.0,V=0.2,R=0.05;

    std::cout<<"S0: "<<S0<<std::endl;
    std::cout<<"K: "<<K<<std::endl;
    std::cout<<"V: "<<V<<std::endl;
    std::cout<<"R: "<<R<<std::endl;
    std::cout<<"T: "<<T<<std::endl;

    std::cout<<"模拟次数："<<simN<<std::endl;
    std::cout<<"时间步数："<<time_step<<std::endl;

    //欧式期权
    Option option(S0,K,T,R,V);
    option.MC_pricing(simN,time_step);
    std::cout<<"看涨期权价格为："<<option.price<<std::endl;
    std::cout<< "欧式期权误差为："<<option.error<<std::endl;
    std::cout << "欧式期权95%置信区间为 ["<< option.price-1.96*option.error<<" ; "<< option.price+1.96*option.error<<" ]"<< std::endl;
    std::cout<<"使用 Euler Maruyama 方法"<<endl;
    std::cout<<std::endl;

    auto end=system_clock::now();
    auto durat=duration_cast<microseconds>(end-start);
    std::cout<<"total cost "<<double(durat.count())*microseconds::period::num/microseconds::period::den<<endl;

    return;
}
int main(int argc, char **argv)
{
    srand(random_seed);
    
    if(argc>=2)
    {
        simN=atoi(argv[1]);
    }
    if(argc>=3)
    {
        time_step=atoi(argv[2]);
    }
    for(int i=0;i<TYPE_NUM;i++)
    {
        test_flag[i]=false;
    }
    ncclComm_t comms[DNUM];
    int devs[DNUM]={0,1};

    test_option(simN,time_step);

    return 0;
}
