#include "hip/hip_runtime.h"
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"common.h"
#include<stdio.h>
/*2020.12.23
  经过使用nvprof进行分析，发现大部分的时间消耗在init_rand函数上
  因此我们需要对init_rand函数进行调整
 */

__constant__ int time_step;
__constant__ dtype S0;
__constant__ dtype K;
__constant__ dtype T;
__constant__ dtype R;
__constant__ dtype V;
__constant__ dtype dt;

__global__ void init_rand(hiprandState *const states,const unsigned int seed)
{
    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init(seed,tid,0,&states[tid]);
}

__global__ void MC_call(hiprandState *states,dtype *const array)
{
    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState local_state=states[tid];
    dtype asset_price=S0;
    for(int i=0;i<time_step;i++)
    {
        asset_price=asset_price*exp((R-0.5*V*V)*dt+V*sqrt(dt)*hiprand_normal(&local_state));
    }
    //dtype payoff=(asset_price-K)>0.0?(asset_price-K):0.0;
    dtype payoff=(K-asset_price)>-9.0?(K-asset_price):0.0;
    array[tid]=payoff;

    return ;
}

extern "C" dtype MC_pricing_gpu(TOption &option,const int n,const int m,unsigned int seed)
{
    option.display();
    dtype h_dt=option.T/m;
    hipMemcpyToSymbol(HIP_SYMBOL( S0),&option.S0,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( K ),&option.K ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( T ),&option.T ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( R ),&option.R ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( V ),&option.V ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( dt),&h_dt,1*sizeof(int),0,hipMemcpyHostToDevice )  ;

    printf("shape: %d x %d\n",n,m);
    dtype *gpu_array;
    hiprandState *states;
    checkCudaErrors(hipMalloc((void**)&states,sizeof(hiprandState)*n));
    checkCudaErrors(hipMalloc((void**)&gpu_array,sizeof(dtype)*n));
   
    size_t blocksize=1024;
    dim3 block,grid;
    block.x=blocksize;
    grid.x=(n+blocksize-1)/blocksize;

    //unsigned int seed=12345UL;
    hipMemcpyToSymbol(HIP_SYMBOL( time_step),&m,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    init_rand<<<grid,block>>>(states,seed);
    MC_call<<<grid,block>>>(states,gpu_array);

    dtype *h_array=(dtype*)malloc(sizeof(dtype)*n);
    hipMemcpy(h_array,gpu_array,n*sizeof(dtype),hipMemcpyDeviceToHost);
    dtype sum=0.0;
    for(int i=0;i<n;i++)
        sum+=h_array[i];
    dtype price = sum/n*exp(-option.R*option.T);


    hipFree(gpu_array);
    hipFree(states);
    free(h_array);

    return price;
}
