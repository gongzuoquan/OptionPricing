#include "hip/hip_runtime.h"
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"option.h"
#include<stdio.h>
/*2020.12.23
  经过使用nvprof进行分析，发现大部分的时间消耗在init_rand函数上
  因此我们需要对init_rand函数进行调整
 */

__constant__ int time_step;
__constant__ dtype S0;
__constant__ dtype K;
__constant__ dtype T;
__constant__ dtype R;
__constant__ dtype V;
__constant__ dtype dt;

__global__ void init_rand(hiprandState *const states,const unsigned int seed)
{
    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init(seed,tid,0,&states[tid]);
}

__global__ void MC_call(hiprandState *states,dtype *const array)
{
    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState local_state=states[tid];
    dtype asset_price=S0;
    for(int i=0;i<time_step;i++)
    {
        asset_price=asset_price*exp((R-0.5*V*V)*dt+V*sqrt(dt)*hiprand_normal(&local_state));
    }
    //dtype payoff=(asset_price-K)>0.0?(asset_price-K):0.0;
    dtype payoff=(K-asset_price)>-9.0?(K-asset_price):0.0;
    array[tid]=payoff;

    return ;
}

void Option::MC_pricing_gpu(const int n,const int m,unsigned int seed)
{
    display();
    dtype h_dt=T/m;
    hipMemcpyToSymbol(HIP_SYMBOL( S0),&S0,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( K ),&K ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( T ),&T ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( R ),&R ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( V ),&V ,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    hipMemcpyToSymbol(HIP_SYMBOL( dt),&h_dt,1*sizeof(int),0,hipMemcpyHostToDevice )  ;

    printf("shape: %d x %d\n",n,m);
    dtype *gpu_array;
    hiprandState *states;
    checkCudaErrors(hipMalloc((void**)&states,sizeof(hiprandState)*n));
    checkCudaErrors(hipMalloc((void**)&gpu_array,sizeof(dtype)*n));
   
    size_t blocksize=1024;
    dim3 block,grid;
    block.x=blocksize;
    grid.x=(n+blocksize-1)/blocksize;

    //unsigned int seed=12345UL;
    hipMemcpyToSymbol(HIP_SYMBOL( time_step),&m,1*sizeof(int),0,hipMemcpyHostToDevice )  ;
    init_rand<<<grid,block>>>(states,seed);
    MC_call<<<grid,block>>>(states,gpu_array);

    dtype *h_array=(dtype*)malloc(sizeof(dtype)*n);
    hipMemcpy(h_array,gpu_array,n*sizeof(dtype),hipMemcpyDeviceToHost);
    dtype sum=0.0;
    for(int i=0;i<n;i++)
        sum+=h_array[i];
    price = sum/n*exp(-R*T);


    hipFree(gpu_array);
    hipFree(states);
    free(h_array);

    return;
}

void Option::display()
{
    printf("S0: %.2f\n",S0);
    printf("K: %.2f\n",K);
    printf("T: %.2f\n",T);
    printf("R: %.2f\n",R);
    printf("V: %.2f\n",V);
    printf("\n");
}
